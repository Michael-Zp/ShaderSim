
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t barycentricCuda(const float3 *v0, const float3 *v1, const float3 *v2, const float *da, const float *db, const float *dc, float *dOut, int2 framebufferSize);

__device__ __inline__ float dot(const float2 a, const float2 b)
{
	return (a.x * b.x) + (a.y * b.y);
}

__device__ float2 calculatePosition(int x, int y, float width, float height)
{
	float2 fragSize = make_float2(2 / width, 2 / height);

	return make_float2(fragSize.x * x + fragSize.y / 2 - 1, (fragSize.y * y + fragSize.y / 2 - 1) * -1);
}

__global__ void baryKernel(const float3 *v0, const float3 *v1, const float3 *v2, const float *da, const float *db, const float *dc, float *dOut, int *width, int *height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < *width && y < *height)
	{
		float2 pos = calculatePosition(x, y, *width, *height);
		float2 t0 = make_float2(v2->x, v2->y);
		float2 t1 = make_float2(v0->x, v0->y);
		float2 t2 = make_float2(v1->x, v1->y);

		float2 v0 = make_float2(t1.x - t0.x, t1.y - t0.y);
		float2 v1 = make_float2(t2.x - t0.x, t2.y - t0.y);
		float2 v2 = make_float2(pos.x - t0.x, pos.y - t0.y);

		float d00 = dot(v0, v0);
		float d01 = dot(v0, v1);
		float d11 = dot(v1, v1);
		float d20 = dot(v2, v0);
		float d21 = dot(v2, v1);
		float denom = d00 * d11 - d01 * d01;

		float baryX = (d11 * d20 - d01 * d21) / denom;
		float baryY = (d00 * d21 - d01 * d20) / denom;
		float baryZ = 1 - baryX - baryY;

		if (baryX > 0 && baryY > 0 && baryZ > 0)
		{
			dOut[y * *width + x] = *da * baryX + *db * baryY + *dc * baryZ;
		}
		else
		{
			dOut[y * *width + x] = 0;
		}
	}


}


int main()
{
	printf("\n\n\nBarycentric:\n");

	int2 framebufferSize = make_int2(50, 50);
	float3 bary_v0 = make_float3(0, 1, 0);
	float3 bary_v1 = make_float3(1, -1, 0);
	float3 bary_v2 = make_float3(-1, -1, 0);
	float bary_da = 3;
	float bary_db = 2;
	float bary_dc = 1;
	float *bary_dOut = (float*)malloc(framebufferSize.x * framebufferSize.y * sizeof(float*));

	// Barycentric in parallel.
	hipError_t cudaStatus = barycentricCuda(&bary_v0, &bary_v1, &bary_v2, &bary_da, &bary_db, &bary_dc, bary_dOut, framebufferSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "barycentricCuda failed!");
		return 1;
	}

	printf("{\n");
	for (int y = 0; y < framebufferSize.y; y++)
	{
		printf("  {");
		for (int x = 0; x < framebufferSize.x; x++)
		{
			printf("%.1f|", bary_dOut[x + y * framebufferSize.y]);
		}
		printf("}\n");
	}
	printf("}\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

#define myMalloc(VAR, SIZE, TYPE) cudaStatus = hipMalloc((void**)&VAR, SIZE * sizeof(TYPE)); \
if (cudaStatus != hipSuccess) {\
		fprintf(stderr, "hipMalloc failed!"); \
		goto Error; \
}

#define myVarOnGPU(VAR, SOURCEVAR, SIZE, TYPE) cudaStatus = hipMalloc((void**)&VAR, SIZE * sizeof(TYPE));\
	if (cudaStatus != hipSuccess) {\
		fprintf(stderr, "hipMalloc failed!");\
		goto Error;\
	}\
	cudaStatus = hipMemcpy(VAR, SOURCEVAR, SIZE * sizeof(TYPE), hipMemcpyHostToDevice);\
	if (cudaStatus != hipSuccess) {\
		fprintf(stderr, "hipMemcpy failed!");\
		goto Error;\
	}


// Helper function for using CUDA to add vectors in parallel.
hipError_t barycentricCuda(const float3 *v0, const float3 *v1, const float3 *v2, const float *da, const float *db, const float *dc, float *dOut, int2 framebufferSize)
{
	int length = framebufferSize.x * framebufferSize.y;
	int bytes = length * sizeof(float);

	const dim3 windowSize(framebufferSize.x, framebufferSize.y);
	const dim3 blockSize(16, 16, 1);
	const dim3 gridSize(windowSize.x / blockSize.x + 1, windowSize.y / blockSize.y + 1);

	

	float3 *dev_v0 = 0;
	float3 *dev_v1 = 0;
	float3 *dev_v2 = 0;
	float *dev_da = 0;
	float *dev_db = 0;
	float *dev_dc = 0;
	struct hipPitchedPtr dstGPU;
	int *dev_width = 0;
	int *dev_height = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate and populate GPU buffers for vectors and data.
	myVarOnGPU(dev_v0, v0, 1, float3);
	myVarOnGPU(dev_v1, v1, 1, float3);
	myVarOnGPU(dev_v2, v2, 1, float3);
	myVarOnGPU(dev_da, da, 1, float);
	myVarOnGPU(dev_db, db, 1, float);
	myVarOnGPU(dev_dc, dc, 1, float);
	myVarOnGPU(dev_width, &framebufferSize.x, 1, int);
	myVarOnGPU(dev_height, &framebufferSize.y, 1, int);
	cudaStatus = hipMalloc3D(&dstGPU, make_hipExtent(framebufferSize.x * sizeof(float), framebufferSize.y, 1));


	// Launch a kernel on the GPU with one thread for each element.
	baryKernel <<<gridSize, blockSize>>> (dev_v0, dev_v1, dev_v2, dev_da, dev_db, dev_dc, (float *)dstGPU.ptr, dev_width, dev_height);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "barycentricCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching barycentricCuda!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dOut, dstGPU.ptr, bytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dstGPU.ptr);
	hipFree(dev_dc);
	hipFree(dev_db);
	hipFree(dev_da);
	hipFree(dev_v2);
	hipFree(dev_v0);
	hipFree(dev_v1);

	return cudaStatus;
}
