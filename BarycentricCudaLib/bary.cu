
#include <hip/hip_runtime.h>
﻿extern "C" {
	//Device code
	__device__ __inline__ float dot(const float2 a, const float2 b)
	{
		return (a.x * b.x) + (a.y * b.y);
	}

	__device__ float2 calculatePosition(int x, int y, float width, float height)
	{
		float2 fragSize = make_float2(2 / width, 2 / height);

		return make_float2(fragSize.x * x + fragSize.y / 2 - 1, fragSize.y * y + fragSize.y / 2 - 1);
	}

	__global__ void baryKernel(const float2 *v0, \
		const float2 *v1, \
		const float2 *v2, \
		const int *dCount, \
		const float *da, \
		const float *db, \
		const float *dc, \
		float *dOut, \
		int *dOut_valid, \
		const int *width, \
		const int *height)
	{
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < *width && y < *height)
		{
			float2 pos = calculatePosition(x, y, *width, *height);
			float2 t0 = make_float2(v2->x, v2->y);
			float2 t1 = make_float2(v0->x, v0->y);
			float2 t2 = make_float2(v1->x, v1->y);

			float2 v0 = make_float2(t1.x - t0.x, t1.y - t0.y);
			float2 v1 = make_float2(t2.x - t0.x, t2.y - t0.y);
			float2 v2 = make_float2(pos.x - t0.x, pos.y - t0.y);

			float d00 = dot(v0, v0);
			float d01 = dot(v0, v1);
			float d11 = dot(v1, v1);
			float d20 = dot(v2, v0);
			float d21 = dot(v2, v1);
			float denom = d00 * d11 - d01 * d01;

			float baryX = (d11 * d20 - d01 * d21) / denom;
			float baryY = (d00 * d21 - d01 * d20) / denom;
			float baryZ = 1 - baryX - baryY;

			if (baryX > 0 && baryY > 0 && baryZ > 0)
			{
				for (int i = 0; i < *dCount; i++)
				{
					dOut[y * *width + x + i * (*width * *height)] = da[i] * baryX + db[i] * baryY + dc[i] * baryZ;
				}
				dOut_valid[y * *width + x] = 1;
			}
			else
			{
				dOut[y * *width + x] = 0;
				dOut_valid[y * *width + x] = 0;
			}
		}

	}
}
