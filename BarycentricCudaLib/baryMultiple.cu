
#include <hip/hip_runtime.h>
﻿extern "C" {
	//Device code
	__device__ __inline__ float dot(const float2 a, const float2 b)
	{
		return (a.x * b.x) + (a.y * b.y);
	}

	__device__ float2 calculatePosition(int x, int y, float width, float height)
	{
		float2 fragSize = make_float2(2 / width, 2 / height);

		return make_float2(fragSize.x * x + fragSize.y / 2 - 1, fragSize.y * y + fragSize.y / 2 - 1);
	}

	__global__ void baryKernel(const float2 *v0, \
		const float2 *v1, \
		const float2 *v2, \
		const unsigned int dCount, \
		const unsigned int primitivesCount, \
		const float *da, \
		const float *db, \
		const float *dc, \
		float *dOut, \
		int *dOut_valid_frament, \
		int *dOut_valid_pixel, \
		const int width, \
		const int height)
	{
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

		if (x < width && y < height && z < primitivesCount)
		{
			float2 pos = calculatePosition(x, y, width, height);
			float2 t0 = make_float2(v2[z].x, v2[z].y);
			float2 t1 = make_float2(v0[z].x, v0[z].y);
			float2 t2 = make_float2(v1[z].x, v1[z].y);

			float2 v0 = make_float2(t1.x - t0.x, t1.y - t0.y);
			float2 v1 = make_float2(t2.x - t0.x, t2.y - t0.y);
			float2 v2 = make_float2(pos.x - t0.x, pos.y - t0.y);

			float d00 = dot(v0, v0);
			float d01 = dot(v0, v1);
			float d11 = dot(v1, v1);
			float d20 = dot(v2, v0);
			float d21 = dot(v2, v1);
			float denom = d00 * d11 - d01 * d01;

			float baryX = (d11 * d20 - d01 * d21) / denom;
			float baryY = (d00 * d21 - d01 * d20) / denom;
			float baryZ = 1 - baryX - baryY;

			int rowSize = width;
			int gridSize = rowSize * height;
			int triangleBlockSize = gridSize * dCount;
			
			int outDataBaseIndex = x + y * rowSize + z * triangleBlockSize;
			int validIndex = x + y * rowSize + z * gridSize;

			if (baryX > 0 && baryY > 0 && baryZ > 0)
			{
				int inDataBaseIndex = z * dCount;
				for (int i = 0; i < dCount; i++)
				{
					int idx = inDataBaseIndex + i;
					dOut[outDataBaseIndex + i * gridSize] = da[idx] * baryX + db[idx] * baryY + dc[idx] * baryZ;
				}
				dOut_valid_frament[validIndex] = 1;
				dOut_valid_pixel[x + y * rowSize] += 1;
			}
			else
			{
				for (int i = 0; i < dCount; i++)
				{
					dOut[outDataBaseIndex + i * gridSize] = 0;
				}
				dOut_valid_frament[validIndex] = 0;
			}
		}

	}
}
