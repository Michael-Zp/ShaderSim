
#include <hip/hip_runtime.h>
﻿extern "C" {
	//Device code
	__device__ __inline__ float dot(const float2 a, const float2 b)
	{
		return (a.x * b.x) + (a.y * b.y);
	}

	__device__ float2 calculatePosition(int x, int y, float width, float height)
	{
		float2 fragSize = make_float2(2 / width, 2 / height);

		return make_float2(fragSize.x * x + fragSize.y / 2 - 1, (fragSize.y * y + fragSize.y / 2 - 1) * -1);
	}

	__global__ void baryKernel(const float3 *a, const float3 *b, const float3 *c, const float *da, const float *db, const float *dc, float *dOut, int *width, int *height)
	{
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < *width && y < *height)
		{
			float2 pos = calculatePosition(x, y, *width, *height);
			float2 t0 = make_float2(c->x, c->y);
			float2 t1 = make_float2(a->x, a->y);
			float2 t2 = make_float2(b->x, b->y);

			float2 v0 = make_float2(t1.x - t0.x, t1.y - t0.y);
			float2 v1 = make_float2(t2.x - t0.x, t2.y - t0.y);
			float2 v2 = make_float2(pos.x - t0.x, pos.y - t0.y);

			float d00 = dot(v0, v0);
			float d01 = dot(v0, v1);
			float d11 = dot(v1, v1);
			float d20 = dot(v2, v0);
			float d21 = dot(v2, v1);
			float denom = d00 * d11 - d01 * d01;

			float baryX = (d11 * d20 - d01 * d21) / denom;
			float baryY = (d00 * d21 - d01 * d20) / denom;
			float baryZ = 1 - baryX - baryY;

			if (baryX > 0 && baryY > 0 && baryZ > 0)
			{
				dOut[y * *width + x] = *da * baryX + *db * baryY + *dc * baryZ;
			}
			else
			{
				dOut[y * *width + x] = 0;
			}
		}

	}
}