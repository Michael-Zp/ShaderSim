
#include <hip/hip_runtime.h>
﻿//Kernel code:
extern "C"  {   
    // Device code
    __global__ void VecAdd(const float* A, const float* B, float* C, int N)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N)
            C[i] = A[i] + B[i];
    }
}